#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <conio.h>

const int arraysize = 3;
hipError_t metodoBusquedaHost(float *valoresintervalo ,float *valoresSplits,unsigned int arraysize);

__device__ float d_valorInicial = 0;
__device__ float d_valorfinal = 20;

float h_valorInicial = 0;
float h_valorfinal = 20;

__global__ void metodoBusqueda(float *vect , float *valorSplits)
{
	int i;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	vect[i] = valorSplits[i] * 2;
}

int main()
{
	float valoresIntervalo [arraysize] = {0};
	float valoresSplits [arraysize] = {0};
	hipError_t cudaStatus = metodoBusquedaHost(valoresIntervalo,valoresSplits,arraysize);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "metodoBusquedaHost fallo!");
		_getch();
		return 1;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset fallo!");
		_getch();
		return 1;
	}

	for (int i = 0; i < 3; i++)
	{
		float number = valoresIntervalo[i];
		printf("%g\n", number); 
	}

	_getch(); 
	return 0;
}

hipError_t metodoBusquedaHost(float *valoresintervalo,float * valoresSplits,unsigned int size)
{
	float *dev_valorinicial = 0;
	float *dev_valorfinal = 0;
	float *dev_valoresintervalos = 0;
	float *dev_valorSplits = 0;

	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_valorinicial,size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc fallo!");
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_valorSplits,size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc fallo!");
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_valorfinal,size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc fallo!");
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_valoresintervalos,size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc fallo!");
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(dev_valoresintervalos, valoresintervalo, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc fallo!");
		return cudaStatus;
	}

	int firstsplitValue = (h_valorfinal - h_valorInicial)/4;
	int secondSplit = (h_valorfinal - h_valorInicial) /2;
	int thirdSplit = firstsplitValue + secondSplit;

	valoresSplits[0] = firstsplitValue;
	valoresSplits[1] = secondSplit;
	valoresSplits[2] =thirdSplit;

	cudaStatus = hipMemcpy(dev_valorSplits, valoresSplits, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc fallo!");
		return cudaStatus;
	}

	metodoBusqueda<<<3,1>>>(dev_valoresintervalos,dev_valorSplits);

	cudaStatus = hipMemcpy(valoresintervalo, dev_valoresintervalos,size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc fallo!");
		return cudaStatus;
	}

	return cudaStatus;
}

